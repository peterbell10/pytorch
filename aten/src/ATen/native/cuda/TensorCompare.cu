#include "hip/hip_runtime.h"
#include <ATen/NativeFunctions.h>
#include <ATen/NumericUtils.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorCompare.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/cuda/CUDAApplyUtils.cuh>


namespace at { namespace native {

namespace {

void where_kernel_impl(TensorIterator &iter, ScalarType condition_type) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBFloat16, kBool, iter.dtype(), "where_cuda", [&] {
    if (condition_type == at::ScalarType::Byte) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (uint8_t cond_val, scalar_t self_val, scalar_t other_val) -> scalar_t {
          return cond_val ? self_val : other_val;
        });
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (bool cond_val, scalar_t self_val, scalar_t other_val) -> scalar_t {
          return cond_val ? self_val : other_val;
        });
    }
  });
}

void isposinf_kernel_impl(TensorIterator &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "isposinf_cuda", [&]() {
    gpu_kernel(
      iter,
      [] GPU_LAMBDA (scalar_t a) -> bool { return a == std::numeric_limits<scalar_t>::infinity(); }
    );
  });
}

void isneginf_kernel_impl(TensorIterator &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "isneginf_cuda", [&]() {
    gpu_kernel(
      iter,
      [] GPU_LAMBDA (scalar_t a) -> bool { return a == -std::numeric_limits<scalar_t>::infinity(); }
    );
  });
}

void clamp_kernel_impl(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_cuda", [&] {
    auto clamp = []GPU_LAMBDA(scalar_t v, scalar_t lower, scalar_t upper) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (at::_isnan(v)) {
        return v;
      } else {
        return ::min(::max(v, lower), upper);
      }
    };

    if (iter.is_cpu_scalar(2) && iter.is_cpu_scalar(3)) {
      const auto lower = iter.scalar_value<scalar_t>(2);
      const auto upper = iter.scalar_value<scalar_t>(3);
      iter.remove_operand(3);
      iter.remove_operand(2);
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
        return clamp(v, lower, upper);
      });
    } else {
      gpu_kernel(iter, clamp);
    }
  });
}

void clamp_min_kernel_impl(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_min_cuda", [&] {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t v, scalar_t lower) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::max(v, lower);
      }
    });
  });
}

void clamp_max_kernel_impl(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_max_cuda", [&] {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t v, scalar_t upper) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::min(v, upper);
      }
    });
  });
}

} // anonymous namespace


REGISTER_DISPATCH(where_kernel, &where_kernel_impl);
REGISTER_DISPATCH(isposinf_stub, &isposinf_kernel_impl);
REGISTER_DISPATCH(isneginf_stub, &isneginf_kernel_impl);
REGISTER_DISPATCH(clamp_stub, &clamp_kernel_impl);
REGISTER_DISPATCH(clamp_min_stub, &clamp_min_kernel_impl);
REGISTER_DISPATCH(clamp_max_stub, &clamp_max_kernel_impl);

}} // namespace at::native
